#include "hip/hip_runtime.h"
#include <algorithm>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>
#include <torch/types.h>
#include <vector>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])
#define MAX_EXP_F32 88.3762626647949f
#define MIN_EXP_F32 -88.3762626647949f
#define SQRT_2_PI M_SQRT2 *M_2_SQRTPI * 0.5f
// to clear the error among self defined gelu and pytorch gelu. Calculate
// $\sqrt{\frac{\pi}{2}}$ by $\sqrt{2 * \pi} / 2$

#define GELU_OPS gelu_tanh_approximate

// There is no half presicion operation like sinh, cosh, tanh. [Half Math
// Functions](https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH____HALF__FUNCTIONS.html#group__CUDA__MATH____HALF__FUNCTIONS)
// $$ tanh(x) = \frac{exp^{2x} - 1}{exp^{2x} + 1}$$
// But ops above will introduce error.
// pytorch transform type while do tanh operator which include in the
// [pytorch/c10/util/BFloat16-math.h](https://github.com/pytorch/pytorch/blob/main/c10/util/BFloat16-math.h)







// __inline__ -> compiler直接把他放到call的地方，而没有真正的call function。
// 1. 减少call 的overhead 2.更多的compiler optimization （CSE，constant folding) 3. 更多的register pressure和code size

__inline__ __device__ float gelu_tanh_approximate(float x) {
  return 0.5f * x * (1.0f + tanhf(SQRT_2_PI * (x + 0.044715f * x * x * x)));
}

__inline__ __device__ float gelu_none_approximate(float x) {
  return x * 0.5 * (1 + erff(x * M_SQRT1_2));
}

// FP32
// GELU tanh approximate: x, y:x 0.5 * x
// * (1.0 + tanh(0.7978845608 * x * (1.0 + 0.044715 * x * x))) grid(N/256),
// block(K=256)
__global__ void gelu_f32_kernel(float *x, float *y, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    float v = fminf(fmaxf(x[idx], MIN_EXP_F32), MAX_EXP_F32);
    y[idx] = GELU_OPS(v);
  }
}

// GELU tanh approximate; Vec4
// grid(N/256), block(256/4)
__global__ void gelu_f32x4_kernel(float *x, float *y, int N) {
  int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  float4 reg_x = FLOAT4(x[idx]);
  float4 reg_y;

  reg_x.x = fminf(fmaxf(reg_x.x, MIN_EXP_F32), MAX_EXP_F32);
  reg_x.y = fminf(fmaxf(reg_x.y, MIN_EXP_F32), MAX_EXP_F32);
  reg_x.z = fminf(fmaxf(reg_x.z, MIN_EXP_F32), MAX_EXP_F32);
  reg_x.w = fminf(fmaxf(reg_x.w, MIN_EXP_F32), MAX_EXP_F32);

  reg_y.x = GELU_OPS(reg_x.x);
  reg_y.y = GELU_OPS(reg_x.y);
  reg_y.z = GELU_OPS(reg_x.z);
  reg_y.w = GELU_OPS(reg_x.w);

  if ((idx + 0) < N) {
    FLOAT4(y[idx]) = reg_y;
  }
}

#define STRINGFY(str) #str
#define TORCH_BINDING_COMMON_EXTENSION(func)                                   \
  m.def(STRINGFY(func), &func, STRINGFY(func));

#define CHECK_TORCH_TENSOR_DTYPE(T, th_type)                                   \
  if (((T).options().dtype() != (th_type))) {                                  \
    std::cout << "Tensor Info:" << (T).options() << std::endl;                 \
    throw std::runtime_error("values must be " #th_type);                      \
  }

#define TORCH_BINDING_GELU(packed_type, th_type, element_type, n_elements)     \
  void gelu_##packed_type(torch::Tensor x, torch::Tensor y) {                  \
    CHECK_TORCH_TENSOR_DTYPE(x, (th_type))                                     \
    CHECK_TORCH_TENSOR_DTYPE(y, (th_type))                                     \
    const int ndim = x.dim();                                                  \
    if (ndim != 2) {                                                           \
      int N = 1;                                                               \
      for (int i = 0; i < ndim; ++i) {                                         \
        N *= x.size(i);                                                        \
      }                                                                        \
      dim3 block(256 / (n_elements));                                          \
      dim3 grid((N + 256 - 1) / 256);                                          \
      gelu_##packed_type##_kernel<<<grid, block>>>(                            \
          reinterpret_cast<element_type *>(x.data_ptr()),                      \
          reinterpret_cast<element_type *>(y.data_ptr()), N);                  \
    } else {                                                                   \
      const int S = x.size(0);                                                 \
      const int K = x.size(1);                                                 \
      const int N = S * K;                                                     \
      if ((K / (n_elements)) <= 1024) {                                        \
        dim3 block(K / (n_elements));                                          \
        dim3 grid(S);                                                          \
        gelu_##packed_type##_kernel<<<grid, block>>>(                          \
            reinterpret_cast<element_type *>(x.data_ptr()),                    \
            reinterpret_cast<element_type *>(y.data_ptr()), N);                \
      } else {                                                                 \
        int N = 1;                                                             \
        for (int i = 0; i < ndim; ++i) {                                       \
          N *= x.size(i);                                                      \
        }                                                                      \
        dim3 block(256 / (n_elements));                                        \
        dim3 grid((N + 256 - 1) / 256);                                        \
        gelu_##packed_type##_kernel<<<grid, block>>>(                          \
            reinterpret_cast<element_type *>(x.data_ptr()),                    \
            reinterpret_cast<element_type *>(y.data_ptr()), N);                \
      }                                                                        \
    }                                                                          \
  }

TORCH_BINDING_GELU(f32, torch::kFloat32, float, 1)
TORCH_BINDING_GELU(f32x4, torch::kFloat32, float, 4)

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  TORCH_BINDING_COMMON_EXTENSION(gelu_f32)
  TORCH_BINDING_COMMON_EXTENSION(gelu_f32x4)
}