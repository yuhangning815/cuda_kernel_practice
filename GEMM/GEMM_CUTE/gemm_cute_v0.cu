#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdlib.h>
#include <cute/tensor.hpp>

template <typename T>
void gen_random_data(T *data, int n);

// Native CUDA GEMM kernel (no CuTe)
template <typename T>
__global__ void gemm_native_cuda(T *C, const T *A, const T *B, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M && col < N) {
        T sum = T(0.0f);
        for (int k = 0; k < K; k++) {
            // A is M x K, B is N x K (so B^T is K x N)
            // C = A * B^T, so C[row][col] = sum(A[row][k] * B[col][k])
            sum += A[row * K + k] * B[col * K + k];
        }
        C[row * N + col] = sum;
    }
}

template <typename T, int kTileM, int kTileN, int kTileK, typename TiledMMA>
__global__ void gemm_simple(T *Cptr, const T *Aptr, const T *Bptr, int m, int n, int k) {

  using namespace cute;

  Tensor A = make_tensor(make_gmem_ptr(Aptr), make_shape(m, k), make_stride(k, Int<1>{}));
  Tensor B = make_tensor(make_gmem_ptr(Bptr), make_shape(n, k), make_stride(k, Int<1>{}));
  Tensor C = make_tensor(make_gmem_ptr(Cptr), make_shape(m, n), make_stride(n, Int<1>{}));

  int ix = blockIdx.x;
  int iy = blockIdx.y;

  // 1. Partition the Global Maqtrix into Tiles 
  Tensor gA = local_tile(A, make_tile(Int<kTileM>{}, Int<kTileK>{}), make_coord(iy, _));
  Tensor gB = local_tile(B, make_tile(Int<kTileN>{}, Int<kTileK>{}), make_coord(ix, _));
  Tensor gC = local_tile(C, make_tile(Int<kTileM>{}, Int<kTileN>{}), make_coord(iy, ix));
  //  gA(kTileM, kTileK, num_tile_k)
  //  gB(kTileN, kTileK, num_tile_k)
  //  gC(kTileM, kTileN) 

  TiledMMA tiled_mma;
  auto thr_mma = tiled_mma.get_slice(threadIdx.x);
  auto tAgA = thr_mma.partition_A(gA);  // (MMA, MMA_M, MMA_K, num_tile_k)
  auto tBgB = thr_mma.partition_B(gB);  // (MMA, MMA_N, MMA_K, num_tile_k)
  auto tCgC = thr_mma.partition_C(gC);  // (MMA, MMA_M, MMA_N)

  // register represetation of the tile 
  auto tArA = thr_mma.partition_fragment_A(gA(_, _, 0));  // (MMA, MMA_M, MMA_K)
  auto tBrB = thr_mma.partition_fragment_B(gB(_, _, 0));  // (MMA, MMA_N, MMA_K)
  auto tCrC = thr_mma.partition_fragment_C(gC(_, _));     // (MMA, MMA_M, MMA_N)
 
  clear(tCrC);
  
  int num_tile_k = size<2>(gA);

  #pragma unroll 1
  for(int itile = 0; itile < num_tile_k; ++itile) {
    cute::copy(tAgA(_, _, _, itile), tArA);
    cute::copy(tBgB(_, _, _, itile), tBrB);

    cute::gemm(tiled_mma, tCrC, tArA, tBrB, tCrC);
  }

  cute::copy(tCrC, tCgC); 
}







int main() {
  srand(10086);

  using T = cute::half_t;
  using namespace cute;

  T *Cptr;
  T *Aptr;
  T *Bptr;

  int m = 81920;
  int n = 256;
  int k = 128;

  hipMalloc(&Cptr, sizeof(T) * m * n);
  hipMalloc(&Aptr, sizeof(T) * m * k);
  hipMalloc(&Bptr, sizeof(T) * k * n);

  T *Aptr_host;
  T *Bptr_host;
  Aptr_host = (T*) malloc(sizeof(T) * m * k);
  Bptr_host = (T*) malloc(sizeof(T) * n * k);
  gen_random_data(Aptr_host, m * k);
  gen_random_data(Bptr_host, n * k);

  hipMemcpy(Aptr, Aptr_host, sizeof(T) * m * k, hipMemcpyHostToDevice);
  hipMemcpy(Bptr, Bptr_host, sizeof(T) * n * k, hipMemcpyHostToDevice);


  // A is row major, B is col major 
  using mma_op = SM80_16x8x16_F16F16F16F16_TN;
  using mma_traits = MMA_Traits<mma_op>;
  using mma_atom = MMA_Atom<mma_traits>;


  // <2,2,1> -> there will be 2 warps horizontal, and 2 warps vertical -> 128 threads per block 
  // <1,2,1> -> each warp takes care of 2 blocks along N. 
  // Data output = <32, 32> , k = 16 
  using MMA = decltype(make_tiled_mma(mma_atom{},   // define the tiled_mma 
                      make_layout(Shape<_2, _2, _1>{}), 
                      make_layout(Shape<_1, _2, _1>{})));

  constexpr int kTileM = 128; 
  constexpr int kTileN = 128; 
  constexpr int kTileK = 32; 

  // Test CuTe implementation
  dim3 block(size(MMA{}));
  dim3 grid(n / kTileN, m / kTileM);
  for (int i = 0; i < 10; ++i) {
    gemm_simple<T, kTileM, kTileN, kTileK, MMA><<<grid, block>>>(Cptr, Aptr, Bptr, m, n, k);
  }

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  printf("CuTe err = %d, str = %s\n", err, hipGetErrorString(err));

  // Test Native CUDA implementation
  T *Cptr_native;
  hipMalloc(&Cptr_native, sizeof(T) * m * n);
  
  dim3 block_native(16, 16);
  dim3 grid_native((n + 15) / 16, (m + 15) / 16);
  for (int i = 0; i < 10; ++i) {
    gemm_native_cuda<T><<<grid_native, block_native>>>(Cptr_native, Aptr, Bptr, m, n, k);
  }

  hipDeviceSynchronize();
  err = hipGetLastError();
  printf("Native CUDA err = %d, str = %s\n", err, hipGetErrorString(err));

  // ---------------------- cublas ----------------------------
  T *Cptr_cublas;

  hipMalloc(&Cptr_cublas, sizeof(T) * m * n);

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  half alpha = half(1.f);
  half beta = half(0.f);
  for (int i = 0; i < 100; ++i) {
    // T = Transpose, N = Normal 
    // in cublas, Ret^T = B^T * A^T (cublas normally return the col major matrix )
    hipblasStatus_t ret = hipblasHgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
          	  n, m, k,
          	  &alpha,
          	  (half *)Bptr, k,
          	  (half *)Aptr, k,
          	  &beta,
          	  (half *)Cptr_cublas, n);       // Cptr_cublas is the result pointer
    if (ret != HIPBLAS_STATUS_SUCCESS) {
      printf("blas err = %d, str = %s\n", ret, cublasGetStatusString(ret));
    }
  }

  hipDeviceSynchronize();
  err = hipGetLastError();
  printf("err = %d, str = %s\n", err, hipGetErrorString(err));

  T *Cptr_host;
  T *Cptr_native_host;
  T *Cptr_cublas_host;

  Cptr_host = (T*)malloc(sizeof(T) * m * n);
  Cptr_native_host = (T*)malloc(sizeof(T) * m * n);
  Cptr_cublas_host = (T*)malloc(sizeof(T) * m * n);

  // compare all three implementations
  hipMemcpy(Cptr_host, Cptr, sizeof(T) * m * n, hipMemcpyDeviceToHost);
  hipMemcpy(Cptr_native_host, Cptr_native, sizeof(T) * m * n, hipMemcpyDeviceToHost);
  hipMemcpy(Cptr_cublas_host, Cptr_cublas, sizeof(T) * m * n, hipMemcpyDeviceToHost);

  float threshold = 0.2;
  int cute_cublas_diff = 0, native_cublas_diff = 0;
  for (int i = 0; i < m * n; ++i) {
    float cute_val = Cptr_host[i];
    float native_val = Cptr_native_host[i];
    float cublas_val = Cptr_cublas_host[i];
    
    if (fabs(cublas_val - cute_val) > threshold) {
      printf("CuTe vs cuBLAS diff: cute=%f, cublas=%f\n", cute_val, cublas_val);
      cute_cublas_diff++;
    }
    if (fabs(cublas_val - native_val) > threshold) {
      printf("Native vs cuBLAS diff: native=%f, cublas=%f\n", native_val, cublas_val);
      native_cublas_diff++;
    }
  }
  
  printf("Differences found: CuTe-cuBLAS: %d, Native-cuBLAS: %d\n", cute_cublas_diff, native_cublas_diff);

  Tensor tensor_C = make_tensor(Cptr_host, make_shape(m, n), make_stride(n, 1));
  Tensor tensor_C_native = make_tensor(Cptr_native_host, make_shape(m, n), make_stride(n, 1));
  Tensor tensor_C_cublas = make_tensor(Cptr_cublas_host, make_shape(m, n), make_stride(n, 1));

  auto tile = make_tile(4, 4);
  auto coor = make_coord(0, 0);
  Tensor tc1 = local_tile(tensor_C, tile, coor);
  Tensor tc1_native = local_tile(tensor_C_native, tile, coor);
  Tensor tc1_cublas = local_tile(tensor_C_cublas, tile, coor);

  printf("CuTe result (first 4x4):\n");
  print_tensor(tc1);
  printf("Native CUDA result (first 4x4):\n");
  print_tensor(tc1_native);
  printf("cuBLAS result (first 4x4):\n");
  print_tensor(tc1_cublas);
}

template <typename T>
void gen_random_data(T *data, int n) {
  for (int i = 0; i < n; ++i) {
    float v = (rand() % 200 - 100) * 0.01;
    data[i] = v;
  }
}