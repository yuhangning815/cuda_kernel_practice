#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>




// 问题：



__global__
void FlashAttention_base_kernel(const float* Q, const float* K, const float* V, const int N, const int d,
                    const int Tc, const int Tr, const int Bc, const int Br, const float softmax_scale,
                    float* l, float *m, float* O) {

    // Q,K,V: [batch_size, num_head, N, d]
    // Scores =  Q · Kᵀ / √dhead                # (B, H, L, L)  
    // Z      =  Attn · V                                 # (B, H, L, dhead)

    // TODO: determine Bc, Br dynamically
    //  K，V的每个分块为 【Bc * d】，Q的每个分块为 【Br * d】
    // blockdim = Bc，一个thread负责一行

    int tx = threadIdx.x;
    int bx = blockIdx.x; int by = blockIdx.y;  // batch and head index

    // Offset into Q,K,V,O,l,m - different for each batch and head
    int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d);  // gridDim.y = num_head
    int lm_offset = (bx * gridDim.y * N) + (by * N);  // offset for l and m

    // Define SRAM for Q,K,V,S
    extern __shared__ float sram[];
    int tile_size = Bc * d;  // size of Qi, Kj, Vj
    float* Qi = sram;
    float* Kj = &sram[tile_size];
    float* Vj = &sram[tile_size * 2];
    float* S = &sram[tile_size * 3];


    // blockdim = Bc, Tc * Bc = N; 一个Block要每Blockdim个行的处理
    for (int j = 0; j < Tc; j++) {

        // Load Kj, Vj to SRAM - 每个thread 一列一列的load；matrix dim = 【Bc, d】 
        // 问题1 ： L2 cache miss：每个thread 之间读的数据差的太远！要within 128/256 Bytes比较好！
        // 问题2： non-coalesced access for GMEM
        for (int x = 0; x < d; x++) {
            Kj[(tx * d) + x] = K[qkv_offset + (tile_size * j) + (tx * d) + x];       
            Vj[(tx * d) + x] = V[qkv_offset + (tile_size * j) + (tx * d) + x];
        }
        __syncthreads();  // such that the inner loop can use the correct Kj, Vj

        for (int i = 0; i < Tr; i++)  {

            // Load Qi to SRAM, l and m to registers
            for (int x = 0; x < d; x++) {
                Qi[(tx * d) + x] = Q[qkv_offset + (tile_size * i) + (tx * d) + x];
            }

            float row_m_prev = m[lm_offset + (Br * i) + tx];   // m = max 
            float row_l_prev = l[lm_offset + (Br * i) + tx];   // l = row_sum

            // S = QK^T, row_m = rowmax(S)
            float row_m = -INFINITY;
            // 做GEMM - 每个Q 行有个thread 要for loop乘 K 的每一行
            for (int y = 0; y < Bc; y++) {
                float sum = 0;
                for (int x = 0; x < d; x++) {
                    sum += Qi[(tx * d) + x] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                S[(Bc * tx) + y] = sum;     // S的size是 【Bc, Bc】，是正确的，不是partial sum

                if (sum > row_m)
                    row_m = sum;
            }

            // P = exp(S - row_m), row_l = rowsum(P)
            float row_l = 0;
            for (int y = 0; y < Bc; y++) {
                S[(Bc * tx) + y] = __expf(S[(Bc * tx) + y] - row_m);   // 问题3： Bank conflict again！if bc 是32的倍数
                row_l += S[(Bc * tx) + y];
            }

            // Compute new m and l
            float row_m_new = max(row_m_prev, row_m);
            float row_l_new = (__expf(row_m_prev - row_m_new) * row_l_prev) + (__expf(row_m - row_m_new) * row_l);

            // Write O, l, m to HBM; 
            // 【Bc, Bc】 * 【Bc, d】 = 【Bc, d】
            for (int x = 0; x < d; x++) {
                float pv = 0;  // Pij * Vj
                for (int y = 0; y < Bc; y++) {
                    pv += S[(Bc * tx) + y] * Vj[(y * d) + x];
                }
                O[qkv_offset + (tile_size * i) + (tx * d) + x] = (1 / row_l_new) \
                    * ((row_l_prev * __expf(row_m_prev - row_m_new) * O[qkv_offset + (tile_size * i) + (tx * d) + x]) \
                    + (__expf(row_m - row_m_new) * pv));
            }
            m[lm_offset + (Br * i) + tx] = row_m_new;
            l[lm_offset + (Br * i) + tx] = row_l_new;
        }
        __syncthreads();  // otherwise, thread can use the wrong Kj, Vj in inner loop
    }
}




torch::Tensor FlashAttention_base_forward(torch::Tensor Q, torch::Tensor K, torch::Tensor V) {
    // Q,K,V: [batch_size, num_head, N, d]

    // TODO: determine Bc, Br dynamically
    //  K，V的每个分块为 【Bc * d】，Q的每个分块为 【Br * d】
    const int Bc = 32; const int Br = 32;

    const int B = Q.size(0); const int nh = Q.size(1);
    const int N = Q.size(2); const int d = Q.size(3);

    const int Tc = ceil((float) N / Bc); const int Tr = ceil((float) N / Br);
    const float softmax_scale = 1.0 / sqrt(d);

    // Initialize O, l, m to HBM
    auto O = torch::zeros_like(Q);
    // 用于 streaming softmax -> 每一行维护一个variable
    auto l = torch::zeros({B, nh, N});
    auto m = torch::full({B, nh, N}, -INFINITY);
    torch::Device device(torch::kCUDA);
    l = l.to(device); m = m.to(device);

    // Calculate SRAM size needed per block
    const int sram_size = (3 * Bc * d * sizeof(float)) + (Bc * Br * sizeof(float));
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);

    dim3 grid_dim(B, nh);  // batch_size x num_heads       -> 一个block处理一个batch的一个head
    dim3 block_dim(Bc);  // Bc threads per block            -> 1D block ！！！

    FlashAttention_base_kernel<<<grid_dim, block_dim, sram_size>>>(
        Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
        N, d, Tc, Tr, Bc, Br, softmax_scale,
        l.data_ptr<float>(), m.data_ptr<float>(), O.data_ptr<float>()
    );
    return O;
}